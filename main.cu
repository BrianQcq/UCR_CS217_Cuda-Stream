/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

#define SegSize 512
#define StreamNum 3
#define BlockSize 512

int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *h_A, *h_B, *h_C;
    float *d_A0, *d_B0, *d_C0;
    float *d_A1, *d_B1, *d_C1;
    float *d_A2, *d_B2, *d_C2;
    size_t A_sz, B_sz, C_sz;
    unsigned VecSize;

    hipStream_t stream0, stream1, stream2;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    if (argc == 1) {
        VecSize = 1000000;
    } else if (argc == 2) {
        VecSize = atoi(argv[1]);   
    } else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    A_sz = VecSize;
    B_sz = VecSize;
    C_sz = VecSize;
    int leftNum = VecSize % (SegSize * StreamNum);
    hipHostAlloc((void**)&h_A, A_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { h_A[i] = (rand()%100)/100.00; }
    hipHostAlloc((void**)&h_B, B_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < B_sz; i++) { h_B[i] = (rand()%100)/100.00; }
    hipHostAlloc((void**)&h_C, C_sz*sizeof(float), hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("size of vector: %u x 1\n  ", VecSize);
    
    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**) &d_A0, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_A1, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_A2, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_B0, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_B1, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_B2, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_C0, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_C1, sizeof(float)*SegSize);
    cuda_ret = hipMalloc((void**) &d_C2, sizeof(float)*SegSize);
    if (cuda_ret != hipSuccess) {
        printf("Fail to hipMalloc on GPU");
        exit(EXIT_FAILURE);
    }
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    // Launch kernel  ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    int i;
    for(i = 0; i < VecSize; i += SegSize * StreamNum)
    {
        hipMemcpyAsync(d_A0, h_A + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_B0, h_B + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_A1, h_A + i + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_B1, h_B + i + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_A2, h_A + i + 2 * SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_B2, h_B + i + 2 * SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream2);

        VecAdd<<<SegSize / BlockSize, BlockSize, 0, stream0>>>(d_A0, d_B0, d_C0, SegSize);
        VecAdd<<<SegSize / BlockSize, BlockSize, 0, stream1>>>(d_A1, d_B1, d_C1, SegSize);
        VecAdd<<<SegSize / BlockSize, BlockSize, 0, stream2>>>(d_A2, d_B2, d_C2, SegSize);

        hipMemcpyAsync(h_C + i, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_C + i + SegSize, d_C1, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(h_C + i + 2 * SegSize, d_C2, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream2);
    }

    // deal with the left data

    i -= SegSize * StreamNum;
    if(leftNum > 2 * SegSize)
    {
        hipMemcpyAsync(d_A0, h_A + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_B0, h_B + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_A1, h_A + i + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_B1, h_B + i + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_A2, h_A + i + 2 * SegSize, (leftNum - 2 * SegSize) * sizeof(float), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_B2, h_B + i + 2 * SegSize, (leftNum - 2 * SegSize) * sizeof(float), hipMemcpyHostToDevice, stream2);

        VecAdd<<<1, BlockSize, 0, stream0>>>(d_A0, d_B0, d_C0, SegSize);
        VecAdd<<<1, BlockSize, 0, stream1>>>(d_A1, d_B1, d_C1, SegSize);
        VecAdd<<<1, leftNum - 2 * SegSize, 0, stream2>>>(d_A2, d_B2, d_C2, leftNum - 2 * SegSize);

        hipMemcpyAsync(h_C + i, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_C + i + SegSize, d_C1, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(h_C + i + 2 * SegSize, d_C2, (leftNum - 2 * SegSize) * sizeof(float), hipMemcpyDeviceToHost, stream2);
    }
    else if(leftNum > SegSize && leftNum <= 2*SegSize)
    {
        hipMemcpyAsync(d_A0, h_A + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_B0, h_B + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_A1, h_A + i + SegSize, (leftNum - SegSize) * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_B1, h_B + i + SegSize, (leftNum - SegSize) * sizeof(float), hipMemcpyHostToDevice, stream1);

        VecAdd<<<1, BlockSize, 0, stream0>>>(d_A0, d_B0, d_C0, SegSize);
        VecAdd<<<1, leftNum - SegSize, 0, stream1>>>(d_A1, d_B1, d_C1, leftNum - SegSize);

        hipMemcpyAsync(h_C + i, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_C + i + SegSize, d_C1, (leftNum - SegSize) * sizeof(float), hipMemcpyDeviceToHost, stream1);
    }
    else if(leftNum > 0 && leftNum <= SegSize)
    {
        hipMemcpyAsync(d_A0, h_A + i, leftNum * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_B0, h_B + i, leftNum * sizeof(float), hipMemcpyHostToDevice, stream0);

        VecAdd<<<1, leftNum, 0, stream0>>>(d_A0, d_B0, d_C0, leftNum);

        hipMemcpyAsync(h_C + i, d_C0, leftNum * sizeof(float), hipMemcpyDeviceToHost, stream0);
    }
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(h_A, h_B, h_C, VecSize);

    // Free memory ------------------------------------------------------------

    hipFree(d_A0);
    hipFree(d_A1);
    hipFree(d_A2);
    hipFree(d_B0);
    hipFree(d_B1);
    hipFree(d_B2);
    hipFree(d_C0);
    hipFree(d_C1);
    hipFree(d_C2);
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
